#include "hip/hip_runtime.h"
#include "stdio.h"
#include "math.h"

__global__ void vecAdd(float* a,float *b, float *c, int n){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i<n){
    c[i] = a[i] + b[i];
  }
  if(i==50){
    printf("GPU working, i=50\n");
  }
}
//multiplies two square matricies together 
__global__ void MatrixMulKernel(float* M, float* N, float* P, int width){
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if((row<width) & (col < width)){
    float Pvalue = 0;
    for(int k =0; k<width; ++k){
      Pvalue += M[row*width+k] * N[k*row+col];
    }
    P[row*Width+col] = Pvalue;
  }
}


int main(){

  int n = 10000;

  size_t bytes = n * sizeof(float);

  float *a_h = (float*)malloc(bytes);
  float *b_h = (float*)malloc(bytes);
  float *c_h = (float*)malloc(bytes);

  float *a_d, *b_d, *c_d;

  hipMalloc(&a_d, bytes);
  hipMalloc(&b_d, bytes);
  hipMalloc(&c_d, bytes);

  for(int i=0; i<n; i++){
    a_h[i] = sin(i) + cos(i);
    b_h[i] = cos(i)*sin(i);
  }

  hipMemcpy(a_d, a_h,bytes, hipMemcpyHostToDevice);
  hipMemcpy(b_d, b_h,bytes, hipMemcpyHostToDevice);

  //createMat<<<ceil(n/256.0), 256>>>(a_d,b_d,c_d, n);
  vecAdd<<<ceil(n/256.0), 256>>>(a_d,b_d,c_d,n);
  hipMemcpy(c_h,c_d, bytes, hipMemcpyDeviceToHost);
  printf("%f",c_h[5]);

  free(a_h);
  free(b_h);
  free(c_h);

  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);

  return 0;
}


