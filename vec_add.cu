
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "math.h"

__global__ void vecAdd(float* a,float *b, float *c, int n){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i<n){
    c[i] = a[i] + b[i];
  }
  if(i==50){
    printf("GPU working, i=50\n");
  }
}

// __global__ void createMat(float* a,float *b,float*c, int n){
//   int i = blockIdx.x * blockDim.x + threadIdx.x;
//   if(i<n){
//     a[i] = sin(i) + cos(i);
//     b[i] = cos(i)*sin(i);
//   }
//   //vecAdd<<<ceil(n/256.0), 256>>>(a,b,c,n);
// }


int main(){

  int n = 10000;

  size_t bytes = n * sizeof(float);

  float *a_h = (float*)malloc(bytes);
  float *b_h = (float*)malloc(bytes);
  float *c_h = (float*)malloc(bytes);

  float *a_d, *b_d, *c_d;

  hipMalloc(&a_d, bytes);
  hipMalloc(&b_d, bytes);
  hipMalloc(&c_d, bytes);

  for(int i=0; i<n; i++){
    a_h[i] = sin(i) + cos(i);
    b_h[i] = cos(i)*sin(i);
  }

  hipMemcpy(a_d, a_h,bytes, hipMemcpyHostToDevice);
  hipMemcpy(b_d, b_h,bytes, hipMemcpyHostToDevice);

  //createMat<<<ceil(n/256.0), 256>>>(a_d,b_d,c_d, n);
  vecAdd<<<ceil(n/256.0), 256>>>(a_d,b_d,c_d,n);
  hipMemcpy(c_h,c_d, bytes, hipMemcpyDeviceToHost);
  printf("%f",c_h[5]);

  free(a_h);
  free(b_h);
  free(c_h);

  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);

  return 0;
}


