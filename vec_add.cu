#include "hip/hip_runtime.h"
#include "stdio.h"
#include "math.h"


__global__ createMat(float* a,float *b,float*c, n){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i<n){
    a[i] = sin(i) + cos(i);
    b[i] = cos(i)*sin(i);
  }
  vecAdd<<<ceil(n/256.0), 256>>>(a,b,c,n);
}

__device__ vecAdd(float* a,float *b, float *c, n){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i<n){
    c[i] = a[i] + b[i];
  }
}

int main(){

int n = 10000;

size_t bytes = n * sizeof(float);

float *a_h = (float*)malloc(bytes);
float *b_h = (float*)malloc(bytes);
float *c_h = (float*)malloc(bytes);

float *a_d, b_d, c_d;

hipMalloc(&a_d, bytes);
hipMalloc(&b_d, bytes);
hipMalloc(&c_d, bytes);

hipMemcpy(a_d, a_h,bytes, hipMemcpyHostToDevice);
hipMemcpy(b_d, b_h,bytes, hipMemcpyHostToDevice);

createMat<<<ceil(n/256.0), 256>>>(a_d,b_d,c_d, n);

hipMemcpy(c_h,c_d, bytes, hipMemcpyDeviceToHost);

free(a_h);
free(b_h);
free(c_h);

hipFree(a_d);
hipFree(b_d);
hipFree(c_d);

return 0;
}


}
