#include "hip/hip_runtime.h"
#include "stdio.h"
#include "math.h"

__device__ void vecAdd(double* a,double *b, double *c, int n){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i<n){
    c[i] = a[i] + b[i];
  }
}

__global__ void createMat(double* a,double *b,double*c, int n){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i<n){
    a[i] = sin(i) + cos(i);
    b[i] = cos(i)*sin(i);
  }
  vecAdd<<<ceil(n/256.0), 256>>>(a,b,c,n);
}



int main(){

int n = 10000;

int bytes = n * sizeof(double);

double *a_h = (double*)malloc(bytes);
double *b_h = (double*)malloc(bytes);
double *c_h = (double*)malloc(bytes);

double *a_d, b_d, c_d;

hipMalloc(&a_d, bytes);
hipMalloc(&b_d, bytes);
hipMalloc(&c_d, bytes);

hipMemcpy(a_d, a_h,bytes, hipMemcpyHostToDevice);
hipMemcpy(b_d, b_h,bytes, hipMemcpyHostToDevice);

createMat<<<ceil(n/256.0), 256>>>(a_d,b_d,c_d, n);

hipMemcpy(c_h,c_d, bytes, hipMemcpyDeviceToHost);

free(a_h);
free(b_h);
free(c_h);

hipFree(a_d);
hipFree(b_d);
hipFree(c_d);

return 0;
}


